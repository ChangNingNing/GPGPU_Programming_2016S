#include "lab1.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
static const unsigned W = 960;
static const unsigned H = 960;
static const unsigned NFRAME = W;

uint32_t V[W];
uint8_t T[H*W*3/2];

typedef struct {
	uint32_t l, r, pivotPos, rPos, pivot;
} Stack;

Stack stack[512];
int stackF = -1;

struct Lab1VideoGenerator::Impl {
	int t = 0;
};

Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
}

Lab1VideoGenerator::~Lab1VideoGenerator() {}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};

void myStackPush(uint32_t l, uint32_t r, uint32_t pivotPos, uint32_t rPos, uint32_t pivot){	
	stackF++;
	stack[stackF].l = l;
	stack[stackF].r = r;
	stack[stackF].pivot = pivot;
	stack[stackF].pivotPos = pivotPos;
	stack[stackF].rPos = rPos;
}

void myPreprocessing(){
	srand(time(NULL));
	for (int i=0; i<W; i++)
		V[i] = i;
	std::random_shuffle(V, V+W);
	myStackPush(0, W-1, 0, W-1, V[0]);
}

void myQuickSort(){
	if (stackF < 0) return;
	unsigned int l, r, pivot, pivotPos, rPos;
	l = stack[stackF].l;
	r = stack[stackF].r;
	pivot = stack[stackF].pivot;
	pivotPos = stack[stackF].pivotPos;
	rPos = stack[stackF].rPos;
	stackF--;

	if(l >= r) return;

	if (pivotPos >= rPos){
		myStackPush(l, pivotPos-1, l, pivotPos-1, V[l]);
		myStackPush(pivotPos+1, r, pivotPos+1, r, V[pivotPos+1]);
		return;
	}
	if (V[pivotPos+1] < pivot){
		V[pivotPos] = V[pivotPos+1];
		V[pivotPos+1] = pivot;
		pivotPos++;
	}
	else{
		int tmp = V[rPos];
		V[rPos] = V[pivotPos+1];
		V[pivotPos+1] = tmp;
		rPos--;
	}

	myStackPush(l, r, pivotPos, rPos, pivot);
}

void Lab1VideoGenerator::Generate(uint8_t *yuv) {
	if (impl->t == 0)
		myPreprocessing();
	else
		for(int i=0; i<13; i++)
			myQuickSort();

	// Render
	for (int i=0; i<H; i++){
		for (int j=0; j<W; j++){
			if (H-1-i > V[j]) T[i*W+j] = 0;
			else T[i*W+j] = 255;
		}
		for (int j=0; j<=stackF; j++)
			if (H-1-i <= stack[j].pivot)
				T[i*W + stack[j].pivotPos] = 76;
	}

	for (int i=0; i<H/2; i++){
		for (int j=0; j<W/2; j++)
			T[W*H + i*W/2 + j] = 128;
		for (int j=0; j<=stackF; j++)
			if (H-1-i*2 <= stack[j].pivot)
				T[W*H + i*W/2 + stack[j].pivotPos/2] = 85;
	}

	for (int i=0; i<H/2; i++){
		for (int j=0; j<W/2; j++)
			T[W*H*5/4 + i*W/2 + j] = 128;
		for (int j=0; j<=stackF; j++)
			if (H-1-i*2 <= stack[j].pivot)
				T[W*H*5/4 + i*W/2 + stack[j].pivotPos/2] = 255;
	}

	hipMemcpy(yuv, T, W*H*3/2, hipMemcpyHostToDevice);
	(impl->t)++;
}
